// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

// This file serve as a simple example for adding a tunable op to onnxruntime.

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <pybind11/pybind11.h>

#include <string>

#include "core/providers/cuda/tunable/cuda_tunable.h"
#include "python/tools/kernel_explorer/kernel_explorer_interface.h"
#include "python/tools/kernel_explorer/kernels/vector_add_kernel.cuh"
#include "contrib_ops/cuda/quantization/matmul_bnb4.cuh"

namespace py = pybind11;

namespace onnxruntime {

// Extend the OpParams so that all specializations have the same parameter passing interface
template <typename T>
struct MatrixFloatBnb4Params : cuda::tunable::OpParams {
  std::string Signature() const override { return std::to_string(n_); }

  T* output_;
  const T* a_;
  const uint8_t* b_;
  const float* scale_;
  const float* quant_map_;
  int m_;
  int n_;
  int k_;
};

template <typename T>
class MatrixFloatBnb4 : public IKernelExplorer {
 public:
  MatrixFloatBnb4(DeviceArray& output,
                  DeviceArray& a,
                  DeviceArray& b,
                  DeviceArray& scale,
                  DeviceArray& quant_map,
                  int m, int n, int k) {
    params_.tuning_ctx = TuningContext();
    params_.stream = Stream();
    params_.output_ = static_cast<T*>(output.ptr());
    params_.a_ = static_cast<T*>(a.ptr());
    params_.b_ = static_cast<uint8_t*>(b.ptr());
    params_.scale_ = static_cast<float*>(scale.ptr());
    params_.quant_map_ = static_cast<float*>(quant_map.ptr());
    params_.m_ = m;
    params_.n_ = n;
    params_.k_ = k;
  }

  void Run() override {
    contrib::cuda::TryMatMulBnb4<T>(
        params_.output_,
        params_.a_,
        params_.b_,
        params_.scale_,
        params_.quant_map_,
        params_.m_,
        params_.n_,
        params_.k_,
        64,
        params_.StreamHandle());
  }

 private:
  // A VectorAddOp<T> is a callable that can process const VectorAddParams<T>*
  using ParamsT = MatrixFloatBnb4Params<T>;
  ParamsT params_{};
};

#define REGISTER_OP(name, type)                                                                             \
  py::class_<name<type>>(m, #name "_" #type)                                                                \
      .def(py::init<DeviceArray&, DeviceArray&, DeviceArray&, DeviceArray&, DeviceArray&, int, int, int>()) \
      .def("SetRepeats", &name<type>::SetRepeats)                                                           \
      .def("Profile", &name<type>::Profile)                                                                 \
      .def("Run", &name<type>::Run);

KE_REGISTER(m) {
  REGISTER_OP(MatrixFloatBnb4, half);
  REGISTER_OP(MatrixFloatBnb4, float);
}

}  // namespace onnxruntime
